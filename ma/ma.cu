#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

// Helper function to perform and time HtoD memory copy
void time_memcpyHtoD(hipDeviceptr_t d_A, float* h_A, size_t bytes) {
    hipEvent_t start, stop;
    hipEventCreateWithFlags(&start, 0);
    hipEventCreateWithFlags(&stop, 0);

    // Record start event
    hipEventRecord(start, 0);

    // Copy from host to device
    hipMemcpyHtoD(d_A, h_A, bytes);

    // Record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for HtoD transfer: %f ms\n", milliseconds);

    // Cleanup events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Helper function to perform and time DtoH memory copy
void time_memcpyDtoH(float* h_A, hipDeviceptr_t d_A, size_t bytes) {
    hipEvent_t start, stop;
    hipEventCreateWithFlags(&start, 0);
    hipEventCreateWithFlags(&stop, 0);

    // Record start event
    hipEventRecord(start, 0);

    // Copy from device to host
    hipMemcpyDtoH(h_A, d_A, bytes);

    // Record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for DtoH transfer: %f ms\n", milliseconds);

    // Cleanup events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t function;
    hipDeviceptr_t d_A;

    // Initialize the CUDA driver API
    hipInit(0);

    // Choose which GPU to use (if any)
    hipDeviceGet(&device, 0);

    // Create a CUDA context for the chosen device
    hipCtxCreate(&context, 0, device);

    size_t N = 2048 * 2048 * 32; // Example size, can be changed as needed
    size_t bytes = N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(bytes);

    // Allocate device memory
    hipMalloc(&d_A, bytes);

    // Perform and time the HtoD memory copy
    time_memcpyHtoD(d_A, h_A, bytes);

    // Perform and time the DtoH memory copy
    time_memcpyDtoH(h_A, d_A, bytes);

    // Cleanup
    hipFree(d_A);
    free(h_A);
    hipCtxDestroy(context);

    return 0;
}