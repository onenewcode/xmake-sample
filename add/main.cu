#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>
#include <random>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
#include <>

#include <stdio.h>

template<class Tdata>
static __device__ void gelu(
    Tdata *__restrict__ a,
    int const * a_s,
    Tdata const *__restrict__ b,
    int const * b_s,
    Tdata  const *__restrict__ c,
    int const * c_s,
    int const conut,
    int const * i_s,
    int const i_s_size){
    // 使用一维grid
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    // 定义c,a,b的偏移
    int c_offset = 0, a_offset = 0, b_offset = 0;

     for (size_t tmp_i = 0; tmp_i < i_s_size; ++i) {
        int k =  i/ i_s[tmp_i];
        c += k * c_s[tmp_i];
        a += k * a_s[tmp_i];
        b += k * b_s[tmp_i];
        i %= i_s[tmp_i];
    }
   *c=*a+*b;
}



int main() {
     const int N = 100;
    size_t bytes = N * sizeof(float);

    // 在主机端分配内存并初始化随机数据
    float* h_input = new float[N];
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(-1.0, 1.0);
    for (int i = 0; i < N; ++i) {
        h_input[i] = distribution(generator);
    }
 h_input[0]=0.036606249235670324f;
    // 打印原始数据
    std::cout << "Original Data:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_input[i] << ",";
    }
    std::cout << std::endl;

    // 分配设备端内存
    half* d_data;
    hipMalloc(&d_data, N * sizeof(half));

    // 将float转换为half并拷贝到设备端
    half* h_half_input = new half[N];
    for (int i = 0; i < N; ++i) {
        h_half_input[i] = __float2half(h_input[i]);
    }
    hipMemcpy(d_data, h_half_input, N * sizeof(half), hipMemcpyHostToDevice);

    // 启动核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    hipDeviceSynchronize();

    // 拷贝结果回主机端
    half* h_half_output = new half[N];
    hipMemcpy(h_half_output, d_data, N * sizeof(half), hipMemcpyDeviceToHost);

    // 将half转换为float并打印输出结果
    std::cout << "Processed Data:" << std::endl;
    for (int i = 0; i < N; ++i) {
        float output = __half2float(h_half_output[i]);
        std::cout << output << " ";
    }
    std::cout << std::endl;

    // 清理
    delete[] h_input;
    delete[] h_half_input;
    delete[] h_half_output;
    hipFree(d_data);

    return 0;
}