#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>
#include <random>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
#include <>

#include <stdio.h>



int main() {
     const int N = 100;
    size_t bytes = N * sizeof(float);

    // 在主机端分配内存并初始化随机数据
    float* h_input = new float[N];
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(-1.0, 1.0);
    for (int i = 0; i < N; ++i) {
        h_input[i] = distribution(generator);
    }
 h_input[0]=0.036606249235670324f;
    // 打印原始数据
    std::cout << "Original Data:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_input[i] << ",";
    }
    std::cout << std::endl;

    // 分配设备端内存
    half* d_data;
    hipMalloc(&d_data, N * sizeof(half));

    // 将float转换为half并拷贝到设备端
    half* h_half_input = new half[N];
    for (int i = 0; i < N; ++i) {
        h_half_input[i] = __float2half(h_input[i]);
    }
    hipMemcpy(d_data, h_half_input, N * sizeof(half), hipMemcpyHostToDevice);

    // 启动核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    hipDeviceSynchronize();

    // 拷贝结果回主机端
    half* h_half_output = new half[N];
    hipMemcpy(h_half_output, d_data, N * sizeof(half), hipMemcpyDeviceToHost);

    // 将half转换为float并打印输出结果
    std::cout << "Processed Data:" << std::endl;
    for (int i = 0; i < N; ++i) {
        float output = __half2float(h_half_output[i]);
        std::cout << output << " ";
    }
    std::cout << std::endl;

    // 清理
    delete[] h_input;
    delete[] h_half_input;
    delete[] h_half_output;
    hipFree(d_data);

    return 0;
}